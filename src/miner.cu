#include "hip/hip_runtime.h"
#include "miner.cuh"
#include <cstdint>
#include <cstring>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <conio.h>  // For _kbhit() and _getch_
#include <errno.h>  // For errno and strerror

// Byte swap functions
__device__ __host__ __inline__ uint32_t swap32(uint32_t val) {
    return ((val & 0x000000ff) << 24) |
           ((val & 0x0000ff00) << 8)  |
           ((val & 0x00ff0000) >> 8)  |
           ((val & 0xff000000) >> 24);
}

// Parse target hash string into Target structure
__host__ Target parse_target_hash(const char* target_str) {
    Target target;
    
    // Convert hex string to bytes, 8 words of 4 bytes each
    for (int i = 0; i < 8; i++) {
        char word[9];  // 8 chars + null terminator
        strncpy(word, target_str + (i * 8), 8);
        word[8] = '\0';
        
        // Convert hex string to uint32_t and store in big-endian format
        uint32_t value;
        sscanf(word, "%x", &value);
        target.words[i] = value;  // Keep in big-endian for comparison
    }
    
    return target;
}

// Convert compact target format to actual target
__host__ Target decode_compact_target(uint32_t compact) {
    Target target = {0};
    int exp = compact >> 24;
    uint32_t mant = compact & 0x007fffff;
    
    // Add the implicit "1" bit
    if (mant > 0) {
        mant |= 0x00800000;
    }
    
    // For exp=0x1d and mant=0x00ffff:
    // Target should be: 0x00ffff0000000000000000000000000000000000000000000000000000000000
    int shift = 8 * (exp - 3);
    int word_idx = shift / 32;
    int bit_shift = shift % 32;
    
    // Place the mantissa in the correct word, keeping big-endian format
    if (bit_shift == 0) {
        target.words[word_idx] = mant;
    } else {
        target.words[word_idx] = mant << bit_shift;
        if (word_idx < 7) {  // Don't overflow array
            target.words[word_idx + 1] = mant >> (32 - bit_shift);
        }
    }
    
    return target;
}

// SHA-256 Constants
__device__ __constant__ uint32_t k[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

// SHA-256 initial state constants
__device__ const uint32_t sha256_init_state[8] = {
    0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
    0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
};

// SHA-256 functions
__device__ uint32_t rotr(uint32_t x, int n) {
    return (x >> n) | (x << (32 - n));
}

__device__ uint32_t ch(uint32_t x, uint32_t y, uint32_t z) {
    return (x & y) ^ (~x & z);
}

__device__ uint32_t maj(uint32_t x, uint32_t y, uint32_t z) {
    return (x & y) ^ (x & z) ^ (y & z);
}

__device__ uint32_t sigma0(uint32_t x) {
    return rotr(x, 2) ^ rotr(x, 13) ^ rotr(x, 22);
}

__device__ uint32_t sigma1(uint32_t x) {
    return rotr(x, 6) ^ rotr(x, 11) ^ rotr(x, 25);
}

__device__ uint32_t gamma0(uint32_t x) {
    return rotr(x, 7) ^ rotr(x, 18) ^ (x >> 3);
}

__device__ uint32_t gamma1(uint32_t x) {
    return rotr(x, 17) ^ rotr(x, 19) ^ (x >> 10);
}

__device__ void sha256_transform(uint32_t* state, const uint32_t* block) {
    uint32_t w[64];
    uint32_t a, b, c, d, e, f, g, h;
    uint32_t t1, t2;

    // Copy block into first 16 words of w - data is already in big-endian
    for (int i = 0; i < 16; i++) {
        w[i] = block[i];
    }

    // Extend the first 16 words into the remaining 48 words w[16..63]
    for (int i = 16; i < 64; i++) {
        uint32_t s0 = rotr(w[i-15], 7) ^ rotr(w[i-15], 18) ^ (w[i-15] >> 3);
        uint32_t s1 = rotr(w[i-2], 17) ^ rotr(w[i-2], 19) ^ (w[i-2] >> 10);
        w[i] = w[i-16] + s0 + w[i-7] + s1;
    }

    // Initialize working variables
    a = state[0];
    b = state[1];
    c = state[2];
    d = state[3];
    e = state[4];
    f = state[5];
    g = state[6];
    h = state[7];

    // Main loop
    for (int i = 0; i < 64; i++) {
        uint32_t S1 = rotr(e, 6) ^ rotr(e, 11) ^ rotr(e, 25);
        uint32_t ch = (e & f) ^ ((~e) & g);
        t1 = h + S1 + ch + k[i] + w[i];
        uint32_t S0 = rotr(a, 2) ^ rotr(a, 13) ^ rotr(a, 22);
        uint32_t maj = (a & b) ^ (a & c) ^ (b & c);
        t2 = S0 + maj;

        h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;
    }

    // Add the compressed chunk to the current hash value
    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
    state[4] += e;
    state[5] += f;
    state[6] += g;
    state[7] += h;
}

__global__ void sha256_gpu(MiningHeader* header, uint8_t* output, Target target, uint32_t* found) {
    uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Create a local copy of the header and update nonce
    MiningHeader local_header = *header;
    local_header.nonce = header->nonce + tid;

    // First SHA-256 hash
    uint32_t state[8];
    uint8_t bytes[64] = {0};  // Changed to byte array
    memcpy(state, sha256_init_state, sizeof(state));
    
    // if (tid == 0) {
    //     printf("\nAttempting new batch with base nonce: %08x\n", header->nonce);
    //     printf("Thread 0 nonce: %08x\n", local_header.nonce);
    // }
    
    // Build block data in big-endian format for SHA-256
    uint32_t pos = 0;
    
    // Hash (variable length) with length prefix
    bytes[pos++] = local_header.hash_length;  // Variable length prefix
    for (int i = 0; i < local_header.hash_length; i++) {
        bytes[pos++] = local_header.hash[i];
    }
    
    // Address1 (20 bytes) with length prefix
    bytes[pos++] = 0x14;  // Length prefix (20)
    for (int i = 0; i < 20; i++) {
        bytes[pos++] = local_header.address1[i];
    }
    
    // Value (block height) - 4 bytes little-endian
    bytes[pos++] = local_header.value & 0xFF;           // Lowest byte
    bytes[pos++] = (local_header.value >> 8) & 0xFF;    // Low byte
    bytes[pos++] = (local_header.value >> 16) & 0xFF;   // High byte
    bytes[pos++] = (local_header.value >> 24) & 0xFF;   // Highest byte
    
    // Address2 (20 bytes) with length prefix
    bytes[pos++] = 0x14;  // Length prefix (20)
    for (int i = 0; i < 20; i++) {
        bytes[pos++] = local_header.address2[i];
    }
    
    // Flag byte - should be 0 or 1
    bytes[pos++] = local_header.flag;
    
    // Timestamp (4 bytes in little-endian)
    bytes[pos++] = local_header.timestamp & 0xFF;
    bytes[pos++] = (local_header.timestamp >> 8) & 0xFF;
    bytes[pos++] = (local_header.timestamp >> 16) & 0xFF;
    bytes[pos++] = (local_header.timestamp >> 24) & 0xFF;
    
    // Nonce (4 bytes in little-endian)
    bytes[pos++] = local_header.nonce & 0xFF;
    bytes[pos++] = (local_header.nonce >> 8) & 0xFF;
    bytes[pos++] = (local_header.nonce >> 16) & 0xFF;
    bytes[pos++] = (local_header.nonce >> 24) & 0xFF;
    
    /*if (tid == 0) {
        printf("\nBlock data before padding:\n");
        printf("Block length: %u bytes\n", pos);
        printf("Raw bytes in hex:\n");
        for (int i = 0; i < pos; i++) {
            printf("%02x", bytes[i]);
            if ((i + 1) % 32 == 0) printf("\n");
        }
        printf("\n\n");
        
        printf("Nonce position (bytes %d-%d): ", pos-4, pos-1);
        for (int i = pos-4; i < pos; i++) {
            printf("%02x", bytes[i]);
        }
        printf(" (nonce value: %08x)\n", local_header.nonce);
    }*/
    
    // Add padding
    uint32_t original_pos = pos;
    bytes[pos++] = 0x80;  // Padding bit
    
    // Fill with zeros until we have room for the length
    while ((pos % 64) != 56) {  // 56 = 64 - 8 (for length)
        bytes[pos++] = 0;
    }
    
    // Add message length in bits as big-endian
    uint64_t total_bits = (uint64_t)original_pos * 8;
    bytes[pos++] = (total_bits >> 56) & 0xFF;
    bytes[pos++] = (total_bits >> 48) & 0xFF;
    bytes[pos++] = (total_bits >> 40) & 0xFF;
    bytes[pos++] = (total_bits >> 32) & 0xFF;
    bytes[pos++] = (total_bits >> 24) & 0xFF;
    bytes[pos++] = (total_bits >> 16) & 0xFF;
    bytes[pos++] = (total_bits >> 8) & 0xFF;
    bytes[pos++] = total_bits & 0xFF;
    
    /*if (tid == 0) {
        printf("Block data after padding (hex):\n");
        for (int i = 0; i < pos; i++) {
            printf("%02x", bytes[i]);
            if ((i + 1) % 64 == 0) printf("\n");
        }
        printf("\n");
        printf("Total length: %d bytes (%d bits)\n", pos, pos * 8);
    }*/
    
    // Process each 64-byte (512-bit) chunk
    for (uint32_t chunk = 0; chunk < pos; chunk += 64) {
        uint32_t w[64] = {0};
        
        // Copy chunk into first 16 words
        for (int i = 0; i < 16; i++) {
            w[i] = (bytes[chunk + i*4] << 24) |
                   (bytes[chunk + i*4 + 1] << 16) |
                   (bytes[chunk + i*4 + 2] << 8) |
                   bytes[chunk + i*4 + 3];
        }
        
        /*if (tid == 0 && chunk == 0) {
            printf("First 16 words of first chunk:\n");
            for (int i = 0; i < 16; i++) {
                printf("%08x ", w[i]);
                if ((i + 1) % 8 == 0) printf("\n");
            }
            printf("\n");
        }*/
        
        sha256_transform(state, w);
    }
    
    /*if (tid == 0) {
        printf("First SHA-256 state:\n");
        for (int i = 0; i < 8; i++) {
            printf("%08x ", state[i]);
        }
        printf("\n");
    }*/
    
    // Second SHA-256 hash
    uint32_t final_state[8];
    memcpy(final_state, sha256_init_state, sizeof(final_state));
    
    // Prepare block for second hash
    uint8_t final_bytes[64] = {0};  // Changed to byte array
    
    // Copy first hash result
    for (int i = 0; i < 8; i++) {
        final_bytes[i*4] = (state[i] >> 24) & 0xFF;
        final_bytes[i*4 + 1] = (state[i] >> 16) & 0xFF;
        final_bytes[i*4 + 2] = (state[i] >> 8) & 0xFF;
        final_bytes[i*4 + 3] = state[i] & 0xFF;
    }
    
    // Add padding for second hash
    final_bytes[32] = 0x80;  // Padding bit after 32 bytes
    // Length is 256 bits = 0x100
    final_bytes[62] = 0x01;  // Upper byte of 0x100
    final_bytes[63] = 0x00;  // Lower byte of 0x100
    
    // Convert to words for transform
    uint32_t final_w[16] = {0};
    for (int i = 0; i < 16; i++) {
        final_w[i] = (final_bytes[i*4] << 24) |
                     (final_bytes[i*4 + 1] << 16) |
                     (final_bytes[i*4 + 2] << 8) |
                     final_bytes[i*4 + 3];
    }
    
    sha256_transform(final_state, final_w);
    
    /*if (tid == 0) {
        printf("Final SHA-256 state (before endian swap):\n");
        for (int i = 0; i < 8; i++) {
            printf("%08x ", final_state[i]);
        }
        printf("\n");
    }*/
    
    // Compare hash with target (both in big-endian)
    bool valid = true;
    /*if (tid == 0) {
        printf("Comparing hash with target:\n");
        printf("Hash:   ");
        for (int i = 7; i >= 0; i--) {
            printf("%08x", swap32(final_state[i]));  // Keep word in little-endian
        }
        printf("\nTarget: ");
        for (int i = 0; i < 8; i++) {
            printf("%08x", target.words[i]);
        }
        printf("\n");
    }*/

    // Compare in Bitcoin's byte order (reversed words, each word in little-endian)
    for (int i = 7; i >= 0 && valid; i--) {
        uint32_t hash_word = swap32(final_state[i]);  // Convert word to little-endian
        uint32_t target_word = target.words[7-i];
        if (hash_word > target_word) {
            valid = false;
        }
        else if (hash_word < target_word) {
            break;
        }
    }

    if (valid) {
        *found = tid;
        // Copy final hash to output in Bitcoin's byte order (reversed words, each word in little-endian)
        for (int i = 0; i < 8; i++) {
            ((uint32_t*)output)[i] = swap32(final_state[7-i]);  // Convert word to little-endian
        }
    }
}

// Hex string to bytes conversion utility
__host__ bool hex_to_bytes(const char* hex_str, uint8_t* bytes, size_t len) {
    if (!hex_str || !bytes || strlen(hex_str) < len * 2) {
        return false;
    }
    for (size_t i = 0; i < len; i++) {
        char hex[3] = {hex_str[i*2], hex_str[i*2+1], 0};
        char* endptr;
        long val = strtol(hex, &endptr, 16);
        if (*endptr != '\0' || val < 0 || val > 255) {
            return false;
        }
        bytes[i] = (uint8_t)val;
    }
    return true;
}

// Save current mining state to a file
bool save_mining_state(const char* filename, const MiningHeader* header, const Target* target) {
    FILE* f = fopen(filename, "wb");
    if (!f) {
        printf("Error: Could not open file %s for writing\n", filename);
        printf("Error code: %d\n", errno);
        printf("Error message: %s\n", strerror(errno));
        return false;
    }

    // Write magic number and version
    const uint32_t MAGIC = 0x4D494E45;  // "MINE"
    const uint32_t VERSION = 1;
    size_t written;
    
    written = fwrite(&MAGIC, sizeof(MAGIC), 1, f);
    if (written != 1) {
        printf("Error writing magic number\n");
        fclose(f);
        return false;
    }
    
    written = fwrite(&VERSION, sizeof(VERSION), 1, f);
    if (written != 1) {
        printf("Error writing version\n");
        fclose(f);
        return false;
    }

    // Write header
    written = fwrite(header, sizeof(MiningHeader), 1, f);
    if (written != 1) {
        printf("Error writing header\n");
        fclose(f);
        return false;
    }

    // Write target
    written = fwrite(target, sizeof(Target), 1, f);
    if (written != 1) {
        printf("Error writing target\n");
        fclose(f);
        return false;
    }

    if (fclose(f) != 0) {
        printf("Error closing file\n");
        return false;
    }
    return true;
}

// Load mining state from a file
bool load_mining_state(const char* filename, MiningHeader* header, Target* target) {
    FILE* f = fopen(filename, "rb");
    if (!f) {
        printf("Error: Could not open file %s for reading\n", filename);
        return false;
    }

    // Read and verify magic number and version
    uint32_t magic, version;
    if (fread(&magic, sizeof(magic), 1, f) != 1 || magic != 0x4D494E45) {
        printf("Error: Invalid state file format\n");
        fclose(f);
        return false;
    }
    if (fread(&version, sizeof(version), 1, f) != 1 || version != 1) {
        printf("Error: Unsupported state file version\n");
        fclose(f);
        return false;
    }

    // Read header
    if (fread(header, sizeof(MiningHeader), 1, f) != 1) {
        printf("Error: Failed to read header from state file\n");
        fclose(f);
        return false;
    }

    // Read target
    if (fread(target, sizeof(Target), 1, f) != 1) {
        printf("Error: Failed to read target from state file\n");
        fclose(f);
        return false;
    }

    fclose(f);
    return true;
}

bool mine_block(MiningHeader* header, Target target, float time_limit) {
    MiningHeader* d_header;
    uint8_t* d_output;
    uint32_t* d_found;
    hipError_t cuda_status;
    bool success = false;
    
    // Allocate device memory
    if ((cuda_status = hipMalloc(&d_header, sizeof(MiningHeader))) != hipSuccess) {
        printf("Error: Failed to allocate device memory for header: %s\n", hipGetErrorString(cuda_status));
        return false;
    }
    
    if ((cuda_status = hipMalloc(&d_output, 32)) != hipSuccess) {
        printf("Error: Failed to allocate device memory for output: %s\n", hipGetErrorString(cuda_status));
        hipFree(d_header);
        return false;
    }
    
    if ((cuda_status = hipMalloc(&d_found, sizeof(uint32_t))) != hipSuccess) {
        printf("Error: Failed to allocate device memory for found flag: %s\n", hipGetErrorString(cuda_status));
        hipFree(d_header);
        hipFree(d_output);
        return false;
    }
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    // Calculate grid dimensions
    int blocks = 8192;  // Adjust based on your GPU
    
    // Print mining parameters
    printf("Starting mining with parameters:\n");
    printf("Threads per block: %d\n", 256);
    printf("Blocks per grid: %d\n", blocks);
    printf("Hashes per launch: %d\n", 256 * blocks);
    
    float elapsed_time = 0;
    uint64_t total_hashes = 0;
    bool interrupted = false;
    
    while (elapsed_time < time_limit && !interrupted) {
        // Check for keyboard input (Windows)
        if (_kbhit()) {
            char c = _getch();
            printf("\nKey pressed: %d\n", (int)c);  // Debug output
            
            // Save state on Ctrl+C (3) or 'q'
            if (c == 3 || c == 'q' || c == 'Q') {
                printf("\n\nMining interrupted. Saving state...\n");
                const char* state_path = "mining_state.bin";  // Try simple path first
                printf("Current working directory: ");
                fflush(stdout);  // Ensure output is shown
                system("cd");    // Print current directory
                
                printf("Attempting to save state to: %s\n", state_path);
                fflush(stdout);  // Ensure output is shown
                
                if (save_mining_state(state_path, header, &target)) {
                    printf("Mining state saved to %s\n", state_path);
                } else {
                    printf("Failed to save mining state to %s\n", state_path);
                    
                    // Try alternate location
                    state_path = "C:/Users/Omer/Documents/Work/Mine/miner/mining_state.bin";
                    printf("Trying alternate path: %s\n", state_path);
                    if (save_mining_state(state_path, header, &target)) {
                        printf("Mining state saved to alternate location: %s\n", state_path);
                    } else {
                        printf("Failed to save to alternate location\n");
                    }
                }
                interrupted = true;
                break;
            }
        }
        
        printf("\rHashes: %llu (%.2f MH/s)", total_hashes, total_hashes / (elapsed_time * 1000000));
        fflush(stdout);
        
        // Reset found flag
        if ((cuda_status = hipMemset(d_found, 0, sizeof(uint32_t))) != hipSuccess) {
            printf("Error: Failed to reset found flag: %s\n", hipGetErrorString(cuda_status));
            break;
        }
        
        // Copy header to device
        if ((cuda_status = hipMemcpy(d_header, header, sizeof(MiningHeader), hipMemcpyHostToDevice)) != hipSuccess) {
            printf("Error: Failed to copy header to device: %s\n", hipGetErrorString(cuda_status));
            break;
        }
        
        // Launch kernel
        sha256_gpu<<<blocks, 256>>>(d_header, d_output, target, d_found);
        
        if ((cuda_status = hipGetLastError()) != hipSuccess) {
            printf("Error: Failed to launch kernel: %s\n", hipGetErrorString(cuda_status));
            break;
        }
        
        // Check if a valid nonce was found
        uint32_t winning_thread;
        if ((cuda_status = hipMemcpy(&winning_thread, d_found, sizeof(uint32_t), hipMemcpyDeviceToHost)) != hipSuccess) {
            printf("Error: Failed to copy found flag: %s\n", hipGetErrorString(cuda_status));
            break;
        }
        
        if (winning_thread != 0) {
            uint32_t winning_nonce = header->nonce + winning_thread;
            uint32_t output_hash[8];
            if ((cuda_status = hipMemcpy(output_hash, d_output, 32, hipMemcpyDeviceToHost)) != hipSuccess) {
                printf("Error: Failed to copy output hash: %s\n", hipGetErrorString(cuda_status));
                break;
            }
            
            printf("\n\n=== Valid Nonce Found! ===\n");
            printf("Nonce (hex): %08x\n", winning_nonce);
            printf("Nonce (decimal): %u\n", winning_nonce);
            printf("Final Hash: ");
            for (int i = 0; i < 8; i++) {
                printf("%08x", output_hash[i]);
            }
            printf("\nTotal hashes tried: %llu\n", total_hashes);
            printf("Time elapsed: %.2f seconds\n", elapsed_time);
            printf("Hash rate: %.2f MH/s\n", total_hashes / (elapsed_time * 1000000));
            printf("========================\n\n");
            
            header->nonce = winning_nonce;
            success = true;
            break;
        }
        
        // Update progress
        total_hashes += 256 * blocks;
        header->nonce += 256 * blocks;
        
        // Update elapsed time
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        elapsed_time /= 1000.0f;  // Convert to seconds
    }
    
    // Cleanup
    hipFree(d_header);
    hipFree(d_output);
    hipFree(d_found);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return success;
}
